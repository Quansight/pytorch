#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THCUNN/common.h>
#include <THC/THCTensor.hpp>

#include <THCUNN/upsampling.h>
#include <THC/THCDeviceTensor.cuh>
#include <THC/THCDeviceTensorUtils.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>

template<typename Dtype, typename Acctype>
__global__ void nearest_neighbor_4d_kernel(
		const int n,
		const THCDeviceTensor<Dtype, 4> data1,
		THCDeviceTensor<Dtype, 4> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int height1 = data1.getSize(2);
  const int width1 = data1.getSize(3);
  const int height2 = data2.getSize(2);
  const int width2 = data2.getSize(3);
  const float height_scale = (float) height1 / (float) height2;
  const float width_scale = (float) width1 / (float) width2;

  if (index < n) {
    const int w2 = index % width2; // 0:width2-1
    const int h2 = index / width2; // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
	for (int c = 0; c < channels; ++c) {
	  const Dtype val = data1[n][c][h1][w1];
	  data2[n][c][h2][w2] = val;
	}
      }
      return;
    }
    //
    const int h1 = nearest_neighbor_compute_source_index(height_scale, h2, height1);
    const int w1 = nearest_neighbor_compute_source_index(width_scale, w2, width1);
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
	const Dtype val = data1[n][c][h1][w1];
	data2[n][c][h2][w2] = val;
      }
    }
  }
}

// Backward operation
template <typename Dtype, typename Acctype>
__global__ void nearest_neighbor_4d_kernel_backward(
		const int n,
		THCDeviceTensor<Dtype, 4> data1,
		const THCDeviceTensor<Dtype, 4> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int height1 = data1.getSize(2);
  const int width1 = data1.getSize(3);
  const int height2 = data2.getSize(2);
  const int width2 = data2.getSize(3);
  const float height_scale = (float) height1 / (float) height2;
  const float width_scale = (float) width1 / (float) width2;

  if (index < n) {
    const int w2 = index % width2; // 0:width2-1
    const int h2 = index / width2; // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
	for (int c = 0; c < channels; ++c) {
	  const Dtype val = data2[n][c][h2][w2];
	  data1[n][c][h1][w1] = val;
	}
      }
      return;
    }
    //
    const int h1 = nearest_neighbor_compute_source_index(height_scale, h2, height1);
    const int w1 = nearest_neighbor_compute_source_index(width_scale, w2, width1);

    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const Dtype d2val = data2[n][c][h2][w2];
        atomicAdd(data1[n][c][h1][w1].data(), d2val);
      }
    }
  }
}


#include <THCUNN/generic/SpatialUpSamplingNearest.cu>
#include <THC/THCGenerateFloatTypes.h>


// ----

#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialUpSamplingNearest.cu"
#else

#include <THCUNN/common.h>
#include "ATen/cuda/HIPContext.h"

static inline void THNN_(SpatialUpSamplingNearest_shapeCheck)
                        (THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         int nBatch, int nChannels,
                         int inputHeight, int inputWidth,
                         int outputHeight, int outputWidth) {
  THArgCheck(inputHeight > 0 && inputWidth > 0
             && outputHeight > 0 && outputWidth > 0, 2,
             "input and output sizes should be greater than 0,"
             " but got input (H: %d, W: %d) output (H: %d, W: %d)",
             inputHeight, inputWidth, outputHeight, outputWidth);
  if (input != NULL) {
     THCUNN_argCheck(state, THTensor_nDimensionLegacyAll(input) == 4, 2, input,
                     "4D input tensor expected but got: %s");
  }

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 4, 0, nBatch);
    THCUNN_check_dim_size(state, gradOutput, 4, 1, nChannels);
    THCUNN_check_dim_size(state, gradOutput, 4, 2, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, 4, 3, outputWidth);
  }
}


void THNN_(SpatialUpSamplingNearest_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
	   int outputHeight,
           int outputWidth)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  int nbatch = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int inputHeight = THCTensor_(size)(state, input, 2);
  int inputWidth  = THCTensor_(size)(state, input, 3);

  THNN_(SpatialUpSamplingNearest_shapeCheck)(state, input, NULL, nbatch, channels,
		  inputHeight, inputWidth,
		  outputHeight, outputWidth);
  THAssert(inputHeight > 0 && inputWidth > 0 && outputHeight > 0 && outputWidth > 0);

  THCTensor_(resize4d)(state, output,
                       THCTensor_(size)(state, input, 0),
                       THCTensor_(size)(state, input, 1),
		       outputHeight,
                       outputWidth);
  THCTensor_(zero)(state, output);

  THCDeviceTensor<scalar_t, 4> idata = toDeviceTensor<scalar_t, 4>(state, input);
  THCDeviceTensor<scalar_t, 4> odata = toDeviceTensor<scalar_t, 4>(state, output);

  const int num_kernels = outputHeight * outputWidth;
  const int num_threads = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = THCState_getCurrentStream(state);
  nearest_neighbor_4d_kernel<scalar_t, accreal> <<<THCCeilDiv(num_kernels, num_threads), num_threads,
	 0, stream>>>(num_kernels, idata, odata);
  THCudaCheck(hipGetLastError());
}



void THNN_(SpatialUpSamplingNearest_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int nbatch,
	   int nchannels,
	   int inputHeight,
	   int inputWidth,
	   int outputHeight,
	   int outputWidth)
{
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THNN_(SpatialUpSamplingNearest_shapeCheck)(state, NULL, gradOutput, nbatch, nchannels,
		  inputHeight, inputWidth, outputHeight, outputWidth);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resize4d)(state, gradInput, nbatch, nchannels, inputHeight, inputWidth);

  THCTensor_(zero)(state, gradInput);
  THCDeviceTensor<scalar_t, 4> data1 = toDeviceTensor<scalar_t, 4>(state, gradInput);
  THCDeviceTensor<scalar_t, 4> data2 = toDeviceTensor<scalar_t, 4>(state, gradOutput);

  const int num_kernels = outputHeight * outputWidth;
  const int num_threads = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = THCState_getCurrentStream(state);

  nearest_neighbor_4d_kernel_backward<scalar_t, accreal> <<<THCCeilDiv(num_kernels, num_threads),
	  num_threads, 0, stream>>>(num_kernels, data1, data2);
  THCudaCheck(hipGetLastError());
  THCTensor_(free)(state, gradOutput);
}

#endif
