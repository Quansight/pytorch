#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <THCUNN/upsampling.h>
#include <THC/THCDeviceTensor.cuh>
#include <THC/THCDeviceTensorUtils.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <TH/THHalf.h>
#include <THCUNN/THCHalfAutoNumerics.cuh>
#include <THC/THCAtomics.cuh>
#include <c10/macros/Macros.h>

template<typename Dtype, typename Acctype>
C10_LAUNCH_BOUNDS(1024)
__global__ void caffe_gpu_interp2_kernel(const int n,
    const Acctype rdepth, const Acctype rheight, const Acctype rwidth, const bool align_corners,
    const THCDeviceTensor<Dtype, 5> data1, THCDeviceTensor<Dtype, 5> data2) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int depth1 = data1.getSize(2);
  const int height1 = data1.getSize(3);
  const int width1 = data1.getSize(4);
  const int depth2 = data2.getSize(2);
  const int height2 = data2.getSize(3);
  const int width2 = data2.getSize(4);

  if (index < n) {
    const int w2 = (index % (height2*width2)) % width2; // 0:width2-1
    const int h2 = (index % (height2*width2)) / width2; // 0:height2-1
    const int t2 = index / (height2*width2);            // 0:depth2-1
    // special case: just copy
    if (depth1 == depth2 && height1 == height2 && width1 == width2) {
      const int t1 = t2;
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data1[n][c][t1][h1][w1];
          data2[n][c][t2][h2][w2] = val;
        }
      }
      return;
    }
    //
    const Acctype t1r = linear_upsampling_compute_source_index<Acctype>(rdepth, t2, align_corners);
    const int t1 = t1r;
    const int t1p = (t1 < depth1 - 1) ? 1 : 0;
    const Acctype t1lambda = t1r - t1;
    const Acctype t0lambda = Acctype(1) - t1lambda;
    //
    const Acctype h1r = linear_upsampling_compute_source_index<Acctype>(rheight, h2, align_corners);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const Acctype h1lambda = h1r - h1;
    const Acctype h0lambda = Acctype(1) - h1lambda;
    //
    const Acctype w1r = linear_upsampling_compute_source_index<Acctype>(rwidth, w2, align_corners);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
        const Acctype val = t0lambda * (h0lambda * (w0lambda * data1[n][c][t1][h1][w1]
                                                  + w1lambda * data1[n][c][t1][h1][w1+w1p])
                                      + h1lambda * (w0lambda * data1[n][c][t1][h1+h1p][w1]
                                                  + w1lambda * data1[n][c][t1][h1+h1p][w1+w1p]))
                          + t1lambda * (h0lambda * (w0lambda * data1[n][c][t1+t1p][h1][w1]
                                                  + w1lambda * data1[n][c][t1+t1p][h1][w1+w1p])
                                      + h1lambda * (w0lambda * data1[n][c][t1+t1p][h1+h1p][w1]
                                                  + w1lambda * data1[n][c][t1+t1p][h1+h1p][w1+w1p]));
        data2[n][c][t2][h2][w2] = ScalarConvert<Acctype, Dtype>::to(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename Dtype, typename Acctype>
C10_LAUNCH_BOUNDS(1024)
__global__ void caffe_gpu_interp2_kernel_backward(const int n,
    const Acctype rdepth, const Acctype rheight, const Acctype rwidth, const bool align_corners,
    THCDeviceTensor<Dtype, 5> data1, const THCDeviceTensor<Dtype, 5> data2){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  const int batchsize = data1.getSize(0);
  const int channels = data1.getSize(1);
  const int depth1 = data1.getSize(2);
  const int height1 = data1.getSize(3);
  const int width1 = data1.getSize(4);
  const int depth2 = data2.getSize(2);
  const int height2 = data2.getSize(3);
  const int width2 = data2.getSize(4);
  if (index < n) {
    const int w2 = (index % (height2*width2)) % width2; // 0:width2-1
    const int h2 = (index % (height2*width2)) / width2; // 0:height2-1
    const int t2 = index / (height2*width2);            // 0:depth2-1
    // special case: just copy
    if (depth1 == depth2 && height1 == height2 && width1 == width2) {
      const int t1 = t2;
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize ; n++){
        for (int c = 0; c < channels; ++c) {
          const Dtype val = data2[n][c][t1][h1][w1];
          data1[n][c][t2][h2][w2] += val;
        }
      }
      return;
    }
    //
    const Acctype t1r = linear_upsampling_compute_source_index<Acctype>(rdepth, t2, align_corners);
    const int t1 = t1r;
    const int t1p = (t1 < depth1 - 1) ? 1 : 0;
    const Acctype t1lambda = t1r - t1;
    const Acctype t0lambda = Acctype(1) - t1lambda;
    //
    const Acctype h1r = linear_upsampling_compute_source_index<Acctype>(rheight, h2, align_corners);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const Acctype h1lambda = h1r - h1;
    const Acctype h0lambda = Acctype(1) - h1lambda;
    //
    const Acctype w1r = linear_upsampling_compute_source_index<Acctype>(rwidth, w2, align_corners);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const Acctype w1lambda = w1r - w1;
    const Acctype w0lambda = Acctype(1) - w1lambda;
    //
    for (int n = 0; n < batchsize ; n++){
      for (int c = 0; c < channels; ++c) {
        const Dtype d2val = data2[n][c][t2][h2][w2];
        atomicAdd(data1[n][c][t1][h1][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h0lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1][h1][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h0lambda * w1lambda * d2val));
        atomicAdd(data1[n][c][t1][h1+h1p][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h1lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1][h1+h1p][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t0lambda * h1lambda * w1lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h0lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h0lambda * w1lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1+h1p][w1].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h1lambda * w0lambda * d2val));
        atomicAdd(data1[n][c][t1+t1p][h1+h1p][w1+w1p].data(),
                  ScalarConvert<Acctype, Dtype>::to(t1lambda * h1lambda * w1lambda * d2val));
      }
    }
  }
  /////////////////////////////////////////////////////////
}


#include <THCUNN/generic/VolumetricUpSamplingTrilinear.cu>
#include <THC/THCGenerateFloatTypes.h>


// --------

#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/VolumetricUpSamplingTrilinear.cu"
#else

#include <THCUNN/upsampling.h>
#include "ATen/cuda/HIPContext.h"

static inline void THNN_(VolumetricUpSamplingTrilinear_shapeCheck)
                        (THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         int nBatch, int nChannels,
                         int inputDepth, int inputHeight, int inputWidth,
                         int outputDepth, int outputHeight, int outputWidth) {
  THArgCheck(inputDepth > 0 && inputHeight > 0 && inputWidth > 0
             && outputDepth && outputHeight > 0 && outputWidth > 0, 2,
             "input and output sizes should be greater than 0,"
             " but got input (D: %d, H: %d, W: %d) output (D: %d, H: %d, W: %d)",
             inputDepth, inputHeight, inputWidth, outputDepth, outputHeight, outputWidth);
  if (input != NULL) {
     THCUNN_argCheck(state, !input->is_empty() && input->dim() == 5, 2, input,
                     "non-empty 5D input tensor expected but got: %s");
  }

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 5, 0, nBatch);
    THCUNN_check_dim_size(state, gradOutput, 5, 1, nChannels);
    THCUNN_check_dim_size(state, gradOutput, 5, 2, outputDepth);
    THCUNN_check_dim_size(state, gradOutput, 5, 3, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, 5, 4, outputWidth);
  }
}

void THNN_(VolumetricUpSamplingTrilinear_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int outputDepth,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  int nbatch = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int inputDepth = THCTensor_(size)(state, input, 2);
  int inputHeight = THCTensor_(size)(state, input, 3);
  int inputWidth = THCTensor_(size)(state, input, 4);
  THNN_(VolumetricUpSamplingTrilinear_shapeCheck)
       (state, input, NULL,
        nbatch, channels,
        inputDepth, inputHeight, inputWidth,
        outputDepth, outputHeight, outputWidth);

  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resize5d)(state, output,
                       THCTensor_(size)(state, input, 0),
                       THCTensor_(size)(state, input, 1),
                       outputDepth, outputHeight, outputWidth);
  THCTensor_(zero)(state, output);
  THCDeviceTensor<scalar_t, 5> idata = toDeviceTensor<scalar_t, 5>(state, input);
  THCDeviceTensor<scalar_t, 5> odata = toDeviceTensor<scalar_t, 5>(state, output);
  THAssert(inputDepth > 0 && inputHeight > 0 && inputWidth > 0 && outputDepth > 0 && outputHeight > 0 && outputWidth > 0);
  const accreal rdepth = linear_upsampling_compute_scale<accreal>(inputDepth, outputDepth, align_corners);
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);
  const int num_kernels = outputDepth * outputHeight * outputWidth;
  const int num_threads =
    at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel<scalar_t, accreal> <<<THCCeilDiv(num_kernels, num_threads), num_threads ,
   0 , stream>>>(num_kernels, rdepth, rheight, rwidth, align_corners, idata, odata);
  THCudaCheck(hipGetLastError());
}


void THNN_(VolumetricUpSamplingTrilinear_updateGradInput)(
           THCState *state,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int nbatch,
           int nchannels,
           int inputDepth,
           int inputHeight,
           int inputWidth,
           int outputDepth,
           int outputHeight,
           int outputWidth,
           bool align_corners)
{
  THNN_(VolumetricUpSamplingTrilinear_shapeCheck)
       (state, NULL, gradOutput,
        nbatch, nchannels,
        inputDepth, inputHeight, inputWidth,
        outputDepth, outputHeight, outputWidth);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  THCTensor_(resize5d)(state, gradInput, nbatch, nchannels, inputDepth, inputHeight, inputWidth);
  THCTensor_(zero)(state, gradInput);
  THCDeviceTensor<scalar_t, 5> data1 = toDeviceTensor<scalar_t, 5>(state, gradInput);
  THCDeviceTensor<scalar_t, 5> data2 = toDeviceTensor<scalar_t, 5>(state, gradOutput);
  const accreal rdepth = linear_upsampling_compute_scale<accreal>(inputDepth, outputDepth, align_corners);
  const accreal rheight = linear_upsampling_compute_scale<accreal>(inputHeight, outputHeight, align_corners);
  const accreal rwidth = linear_upsampling_compute_scale<accreal>(inputWidth, outputWidth, align_corners);
  const int num_kernels = outputDepth * outputHeight * outputWidth;
  const int num_threads =
    at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = THCState_getCurrentStream(state);
  caffe_gpu_interp2_kernel_backward<scalar_t ,accreal> <<<THCCeilDiv(num_kernels, num_threads),
  num_threads, 0, stream>>>(num_kernels, rdepth, rheight, rwidth, align_corners, data1, data2);
  THCudaCheck(hipGetLastError());
  THCTensor_(free)(state, gradOutput);
}

#endif
